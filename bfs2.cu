#include "hip/hip_runtime.h"
#include "helper.h"
#include <stdio.h>
#include <vector>
#include <limits>
#include <iostream>

#define CUDA_CHECK(cudaStatus)                                      \
    if(cudaStatus != hipSuccess)                                   \
        std::cout << hipGetErrorString(cudaStatus) << std::endl;   \


__global__ void BFS_step2(
    unsigned long long int* d_edges,
    unsigned long long int* d_indices,
    unsigned long long int* d_weights,
    unsigned long long int* d_distances,
    unsigned long long int* d_frontier,
    unsigned long long int* d_frontier_size,
    int* d_is_in_frontier
) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if(id >= *d_frontier_size) {
        return;
    }

    unsigned long long int nodeA = d_frontier[id];

    __syncthreads();
    *d_frontier_size = 0;
    __syncthreads();

    for (unsigned long long int i = d_indices[nodeA]; i < d_indices[nodeA + 1]; i++) {
        unsigned long long int nodeB = d_edges[i];
        unsigned long long int weight = d_weights[i];
        unsigned long long int old_distance = d_distances[nodeB];
        unsigned long long int new_distance = d_distances[nodeA] + weight;
        if (new_distance < old_distance) {
            atomicMin(&d_distances[nodeB], new_distance);
            unsigned long long int is_in_frontier = atomicCAS(&d_is_in_frontier[nodeB], 0, 1);
            if (is_in_frontier == 0) {
                unsigned long long int new_frontier_index = atomicAdd(d_frontier_size, 1);
                d_frontier[new_frontier_index] = nodeB;
            }
        }
    }   
}


std::vector<unsigned long long int> BFS2(
    std::vector<std::vector<unsigned long long int>> edges,
    std::vector<std::vector<unsigned long long int>> weights
) {

    std::vector<unsigned long long int> h_edges;
    std::vector<unsigned long long int> h_indices;
    std::vector<unsigned long long int> h_weights;

    // this can be done in parallel probably
    unsigned long long int current_index = 0;
    for (const auto& inner : edges) {
        h_indices.push_back(current_index);
        h_edges.insert(h_edges.end(), inner.begin(), inner.end());
        current_index += inner.size();
    }
    for (const auto& inner : weights) {
        h_weights.insert(h_weights.end(), inner.begin(), inner.end());
    }
    unsigned long long int n = edges.size(); //number of nodes
    unsigned long long int m = h_edges.size(); //number of edges
    h_indices.push_back(m);
    

    unsigned long long int* d_edges;
    unsigned long long int* d_indices;
    unsigned long long int* d_weights;

    CUDA_CHECK(hipMalloc(&d_edges, h_edges.size() * sizeof(unsigned long long int)));
    CUDA_CHECK(hipMalloc(&d_indices, h_indices.size() * sizeof(unsigned long long int)));
    CUDA_CHECK(hipMalloc(&d_weights, h_weights.size() * sizeof(unsigned long long int)));

    CUDA_CHECK(hipMemcpy(d_edges, h_edges.data(), h_edges.size() * sizeof(unsigned long long int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_indices, h_indices.data(), h_indices.size() * sizeof(unsigned long long int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_weights, h_weights.data(), h_weights.size() * sizeof(unsigned long long int), hipMemcpyHostToDevice));
    unsigned long long int* h_distances = new unsigned long long int[n];
    for (unsigned long long int i = 0; i < n; i++) {
        h_distances[i] = std::numeric_limits<unsigned long long int>::max();
    }
    h_distances[0] = 0;
    unsigned long long int* d_distances;
    CUDA_CHECK(hipMalloc(&d_distances, n * sizeof(unsigned long long int)));
    CUDA_CHECK(hipMemcpy(d_distances, h_distances, n * sizeof(unsigned long long int), hipMemcpyHostToDevice));

    unsigned long long int* d_frontier;
    CUDA_CHECK(hipMalloc(&d_frontier, n * sizeof(unsigned long long int)));
    CUDA_CHECK(hipMemset(d_frontier, 0, n * sizeof(unsigned long long int)));

    unsigned long long int frontier_size = 1; // or may be different value in case of starting
    unsigned long long int* d_frontier_size;
    CUDA_CHECK(hipMalloc(&d_frontier_size, sizeof(unsigned long long int)));
    CUDA_CHECK(hipMemcpy(d_frontier_size, &frontier_size, sizeof(unsigned long long int), hipMemcpyHostToDevice));

   
    int* is_in_frontier;
    CUDA_CHECK(hipMalloc(&is_in_frontier, n * sizeof(int)));
    CUDA_CHECK(hipMemset(is_in_frontier, 0, n * sizeof(int)));

    while(true) {
        CUDA_CHECK(hipMemcpy(&frontier_size, d_frontier_size, sizeof(unsigned long long int), hipMemcpyDeviceToHost));
        if (frontier_size == 0) {
            break;
        }
        CUDA_CHECK(hipMemset(is_in_frontier, 0, n * sizeof(int)));
        BFS_step2<<<(frontier_size + 255) / 256, 256>>>(
            d_edges,
            d_indices,
            d_weights,
            d_distances,
            d_frontier,
            d_frontier_size,
            is_in_frontier
        );
    //endloop
    CUDA_CHECK(hipDeviceSynchronize());
    }

    //copying back the distances
    CUDA_CHECK(hipMemcpy(h_distances, d_distances, n * sizeof(unsigned long long int), hipMemcpyDeviceToHost));
    
    std::vector<unsigned long long int> distances(h_distances, h_distances + n);

    hipFree(d_edges);
    hipFree(d_indices);
    hipFree(d_weights);
    hipFree(d_distances);
    hipFree(d_frontier);
    hipFree(d_frontier_size);
    hipFree(is_in_frontier);

    return distances;

}