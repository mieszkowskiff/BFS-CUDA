#include "hip/hip_runtime.h"
#include "helper.h"
#include <stdio.h>
#include <vector>
#include <limits>
#include <iostream>
#include <chrono>

#define CUDA_CHECK(cudaStatus)                                      \
    if(cudaStatus != hipSuccess)                                   \
        std::cout << hipGetErrorString(cudaStatus) << std::endl;   \


__global__ void BFS_step2(
    unsigned long long int* d_edges,
    unsigned long long int* d_indices,
    unsigned long long int* d_weights,
    unsigned long long int* d_distances,
    unsigned long long int* d_frontier,
    unsigned long long int* d_frontier_size,
    int* d_is_in_frontier
) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if(id >= *d_frontier_size) {
        return;
    }

    unsigned long long int nodeA = d_frontier[id];

    __syncthreads();
    *d_frontier_size = 0;
    __syncthreads();

    for (unsigned long long int i = d_indices[nodeA]; i < d_indices[nodeA + 1]; i++) {
        unsigned long long int nodeB = d_edges[i];
        unsigned long long int weight = d_weights[i];
        unsigned long long int old_distance = d_distances[nodeB];
        unsigned long long int new_distance = d_distances[nodeA] + weight;
        if (new_distance < old_distance) {
            atomicMin(&d_distances[nodeB], new_distance);
            unsigned long long int is_in_frontier = atomicCAS(&d_is_in_frontier[nodeB], 0, 1);
            if (is_in_frontier == 0) {
                unsigned long long int new_frontier_index = atomicAdd(d_frontier_size, 1);
                d_frontier[new_frontier_index] = nodeB;
            }
        }
    }   
}


std::vector<unsigned long long int> BFS2(
    std::vector<std::vector<unsigned long long int>> edges,
    std::vector<std::vector<unsigned long long int>> weights
) {

    std::vector<unsigned long long int> h_edges;
    std::vector<unsigned long long int> h_indices;
    std::vector<unsigned long long int> h_weights;

    // flatten edges and weights
    unsigned long long int current_index = 0;
    for (const auto& inner : edges) {
        h_indices.push_back(current_index);
        h_edges.insert(h_edges.end(), inner.begin(), inner.end());
        current_index += inner.size();
    }
    for (const auto& inner : weights) {
        h_weights.insert(h_weights.end(), inner.begin(), inner.end());
    }
    unsigned long long int n = edges.size();
    unsigned long long int m = h_edges.size();
    h_indices.push_back(m);
    

    unsigned long long int* d_edges;
    unsigned long long int* d_indices;
    unsigned long long int* d_weights;


    // initialize distances
    unsigned long long int* h_distances = new unsigned long long int[n];
    for (unsigned long long int i = 0; i < n; i++) {
        h_distances[i] = std::numeric_limits<unsigned long long int>::max();
    }
    h_distances[0] = 0;

    unsigned long long int* d_distances;
    
    // initialize frontier
    unsigned long long int* d_frontier;

    unsigned long long int frontier_size = 1;
    unsigned long long int* d_frontier_size;
    
    int* is_in_frontier;

    auto start_copying = std::chrono::high_resolution_clock::now();

    // copy data to GPU
    CUDA_CHECK(hipMalloc(&d_edges, h_edges.size() * sizeof(unsigned long long int)));
    CUDA_CHECK(hipMalloc(&d_indices, h_indices.size() * sizeof(unsigned long long int)));
    CUDA_CHECK(hipMalloc(&d_weights, h_weights.size() * sizeof(unsigned long long int)));

    CUDA_CHECK(hipMemcpy(
        d_edges, 
        h_edges.data(), 
        h_edges.size() * sizeof(unsigned long long int), 
        hipMemcpyHostToDevice
        ));
    CUDA_CHECK(hipMemcpy(
        d_indices, 
        h_indices.data(), 
        h_indices.size() * sizeof(unsigned long long int), 
        hipMemcpyHostToDevice
        ));
    CUDA_CHECK(hipMemcpy(
        d_weights, 
        h_weights.data(), 
        h_weights.size() * sizeof(unsigned long long int), 
        hipMemcpyHostToDevice
        ));

    CUDA_CHECK(hipMalloc(&d_distances, n * sizeof(unsigned long long int)));
    CUDA_CHECK(hipMemcpy(d_distances, h_distances, n * sizeof(unsigned long long int), hipMemcpyHostToDevice));

    CUDA_CHECK(hipMalloc(&d_frontier, n * sizeof(unsigned long long int)));
    CUDA_CHECK(hipMemset(d_frontier, 0, n * sizeof(unsigned long long int)));

    CUDA_CHECK(hipMalloc(&d_frontier_size, sizeof(unsigned long long int)));
    CUDA_CHECK(hipMemcpy(d_frontier_size, &frontier_size, sizeof(unsigned long long int), hipMemcpyHostToDevice));

    CUDA_CHECK(hipMalloc(&is_in_frontier, n * sizeof(int)));
    CUDA_CHECK(hipMemset(is_in_frontier, 0, n * sizeof(int)));

    auto end_copying = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed_seconds_copying = end_copying - start_copying;
    std::cout << "Time to copy data to GPU: " << elapsed_seconds_copying.count() << "s\n";

    auto start_processing = std::chrono::high_resolution_clock::now();

    while(true) {
        CUDA_CHECK(hipMemcpy(&frontier_size, d_frontier_size, sizeof(unsigned long long int), hipMemcpyDeviceToHost));
        if (frontier_size == 0) {
            break;
        }

        //bfs step
        CUDA_CHECK(hipMemset(is_in_frontier, 0, n * sizeof(int)));
        BFS_step2<<<(frontier_size + 255) / 256, 256>>>(
            d_edges,
            d_indices,
            d_weights,
            d_distances,
            d_frontier,
            d_frontier_size,
            is_in_frontier
        );
    //endloop
    CUDA_CHECK(hipDeviceSynchronize());
    }

    auto end_processing = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed_seconds_processing = end_processing - start_processing;
    std::cout << "Time to process data on GPU: " << elapsed_seconds_processing.count() << "s\n";

    auto start_copying_back = std::chrono::high_resolution_clock::now();
    CUDA_CHECK(hipMemcpy(h_distances, d_distances, n * sizeof(unsigned long long int), hipMemcpyDeviceToHost));

    auto end_copying_back = std::chrono::high_resolution_clock::now();

    std::chrono::duration<double> elapsed_seconds_copying_back = end_copying_back - start_copying_back;
    std::cout << "Time to copy data back to CPU: " << elapsed_seconds_copying_back.count() << "s\n";
    std::cout << "Total bfs2 time: " << 
        elapsed_seconds_copying.count() + 
        elapsed_seconds_processing.count() + 
        elapsed_seconds_copying_back.count() << "s\n";
    
    std::vector<unsigned long long int> distances(h_distances, h_distances + n);

    hipFree(d_edges);
    hipFree(d_indices);
    hipFree(d_weights);
    hipFree(d_distances);
    hipFree(d_frontier);
    hipFree(d_frontier_size);
    hipFree(is_in_frontier);

    return distances;

}