#include "hip/hip_runtime.h"
#include "helper.h"
#include <stdio.h>
#include <vector>
#include <limits>
#include <iostream>
#include <chrono>

#define CUDA_CHECK(cudaStatus)                                      \
    if(cudaStatus != hipSuccess)                                   \
        std::cout << hipGetErrorString(cudaStatus) << std::endl;   \


__global__ void BFS_step1(
    unsigned long long int* d_edges,
    unsigned long long int* d_indices,
    unsigned long long int* d_weights,
    unsigned long long int* d_distances,
    unsigned long long int* d_frontier,
    unsigned long long int* d_frontier_size,
    unsigned long long int* d_frontier_max_degree,
    int* d_is_in_frontier
) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if(id >= *d_frontier_size * *d_frontier_max_degree) {
        return;
    }
    unsigned long long int frontier_size = *d_frontier_size;
    unsigned long long int location_in_frontier = id % frontier_size;
    unsigned long long int neighbour_index = id / frontier_size;

    unsigned long long int nodeA = d_frontier[location_in_frontier];

    unsigned long long int edge_index = d_indices[nodeA] + neighbour_index;
    if(edge_index >= d_indices[nodeA + 1]) {
        return;
    }
    
    unsigned long long int nodeB = d_edges[edge_index];
    unsigned long long int weight = d_weights[edge_index];

    unsigned long long int new_distance = d_distances[nodeA] + weight;
    unsigned long long int old_distance = d_distances[nodeB];
    __syncthreads();
    *d_frontier_size = 0;
    *d_frontier_max_degree = 0;
    __syncthreads();
    
    if (new_distance < old_distance) {
        atomicMin(&d_distances[nodeB], new_distance);
        
        unsigned long long int is_in_frontier = atomicCAS(&d_is_in_frontier[nodeB], 0, 1);
        
        if (is_in_frontier == 0) {
            
            unsigned long long int new_frontier_index = atomicAdd(d_frontier_size, 1);
            
            d_frontier[new_frontier_index] = nodeB;
            atomicMax(d_frontier_max_degree, d_indices[nodeB + 1] - d_indices[nodeB]);
            
        }
    }
}


std::vector<unsigned long long int> BFS1(
    std::vector<std::vector<unsigned long long int>> edges,
    std::vector<std::vector<unsigned long long int>> weights
) {

    int device = 0;
    hipSetDevice(device);

    // Sprawdzanie dostępnej pamięci globalnej
    size_t freeMem, totalMem;
    hipMemGetInfo(&freeMem, &totalMem);

    std::cout << "Całkowita pamięć GPU: " << totalMem / (1024 * 1024) << " MB" << std::endl;
    std::cout << "Dostępna pamięć GPU: " << freeMem / (1024 * 1024) << " MB" << std::endl;


    std::vector<unsigned long long int> h_edges;
    std::vector<unsigned long long int> h_indices;
    std::vector<unsigned long long int> h_weights;

    // this can be done in parallel probably
    unsigned long long int current_index = 0;
    for (const auto& inner : edges) {
        h_indices.push_back(current_index);
        h_edges.insert(h_edges.end(), inner.begin(), inner.end());
        current_index += inner.size();
    }
    for (const auto& inner : weights) {
        h_weights.insert(h_weights.end(), inner.begin(), inner.end());
    }
    unsigned long long int n = edges.size(); //number of nodes
    unsigned long long int m = h_edges.size(); //number of edges
    h_indices.push_back(m);
    

    unsigned long long int* d_edges;
    unsigned long long int* d_indices;
    unsigned long long int* d_weights;

    
    unsigned long long int* h_distances = new unsigned long long int[n];
    for (unsigned long long int i = 0; i < n; i++) {
        h_distances[i] = std::numeric_limits<unsigned long long int>::max();
    }
    h_distances[0] = 0;
    unsigned long long int* d_distances;


    unsigned long long int* d_frontier;

    unsigned long long int frontier_size = 1; // or may be different value in case of starting
    unsigned long long int* d_frontier_size;
    

    unsigned long long int h_frontier_max_degree = h_indices[1] - h_indices[0];
    // printing indices
    unsigned long long int* d_frontier_max_degree;
    

    int* is_in_frontier;

    auto start_copying = std::chrono::high_resolution_clock::now();

    CUDA_CHECK(hipMalloc(&d_edges, h_edges.size() * sizeof(unsigned long long int)));
    CUDA_CHECK(hipMalloc(&d_indices, h_indices.size() * sizeof(unsigned long long int)));
    CUDA_CHECK(hipMalloc(&d_weights, h_weights.size() * sizeof(unsigned long long int)));

    CUDA_CHECK(hipMemcpy(
        d_edges, 
        h_edges.data(), 
        h_edges.size() * sizeof(unsigned long long int), 
        hipMemcpyHostToDevice
        ));
    CUDA_CHECK(hipMemcpy(
        d_indices, 
        h_indices.data(), 
        h_indices.size() * sizeof(unsigned long long int), 
        hipMemcpyHostToDevice
        ));
    CUDA_CHECK(hipMemcpy(
        d_weights, 
        h_weights.data(), 
        h_weights.size() * sizeof(unsigned long long int), 
        hipMemcpyHostToDevice
        ));
    CUDA_CHECK(hipMalloc(&d_distances, n * sizeof(unsigned long long int)));
    CUDA_CHECK(hipMemcpy(d_distances, h_distances, n * sizeof(unsigned long long int), hipMemcpyHostToDevice));

    CUDA_CHECK(hipMalloc(&d_frontier, n * sizeof(unsigned long long int)));
    CUDA_CHECK(hipMemset(d_frontier, 0, n * sizeof(unsigned long long int)));

    CUDA_CHECK(hipMalloc(&d_frontier_size, sizeof(unsigned long long int)));
    CUDA_CHECK(hipMemcpy(d_frontier_size, &frontier_size, sizeof(unsigned long long int), hipMemcpyHostToDevice));

    CUDA_CHECK(hipMalloc(&d_frontier_max_degree, sizeof(unsigned long long int)));
    CUDA_CHECK(hipMemcpy(
        d_frontier_max_degree, 
        &h_frontier_max_degree, 
        sizeof(unsigned long long int), 
        hipMemcpyHostToDevice
        ));

    CUDA_CHECK(hipMalloc(&is_in_frontier, n * sizeof(int)));
    CUDA_CHECK(hipMemset(is_in_frontier, 0, n * sizeof(int)));

    auto end_copying = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed_seconds_copying = end_copying - start_copying;
    std::cout << "Time to copy data to GPU: " << elapsed_seconds_copying.count() << "s\n";

    
    auto start_processing = std::chrono::high_resolution_clock::now();
    while(true) {
        CUDA_CHECK(hipMemcpy(
            &h_frontier_max_degree, 
            d_frontier_max_degree, 
            sizeof(unsigned long long int), 
            hipMemcpyDeviceToHost
            ));
        CUDA_CHECK(hipMemcpy(
            &frontier_size, 
            d_frontier_size, 
            sizeof(unsigned long long int), 
            hipMemcpyDeviceToHost
            ));
        if (frontier_size == 0) {
            break;
        }
        CUDA_CHECK(hipMemset(is_in_frontier, 0, n * sizeof(int)));
        BFS_step1<<<(frontier_size * h_frontier_max_degree + 255) / 256, 256>>>(
            d_edges,
            d_indices,
            d_weights,
            d_distances,
            d_frontier,
            d_frontier_size,
            d_frontier_max_degree,
            is_in_frontier
        );
    //endloop
    CUDA_CHECK(hipDeviceSynchronize());
    }

    auto end_processing = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed_seconds_processing = end_processing - start_processing;
    std::cout << "Time to process data on GPU: " << elapsed_seconds_processing.count() << "s\n";

    

    auto start_copying_back = std::chrono::high_resolution_clock::now();
    CUDA_CHECK(hipMemcpy(h_distances, d_distances, n * sizeof(unsigned long long int), hipMemcpyDeviceToHost));
    auto end_copying_back = std::chrono::high_resolution_clock::now();

    std::chrono::duration<double> elapsed_seconds_copying_back = end_copying_back - start_copying_back;
    std::cout << "Time to copy data back to CPU: " << elapsed_seconds_copying_back.count() << "s\n";

    
    std::cout << "Total bfs1 time: " << 
        elapsed_seconds_copying.count() + 
        elapsed_seconds_processing.count() + 
        elapsed_seconds_copying_back.count() << 
        "s\n";
    
    std::vector<unsigned long long int> distances(h_distances, h_distances + n);

    hipFree(d_edges);
    hipFree(d_indices);
    hipFree(d_weights);
    hipFree(d_distances);
    hipFree(d_frontier);
    hipFree(d_frontier_size);
    hipFree(d_frontier_max_degree);
    hipFree(is_in_frontier);

    return distances;

}